#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;


__device__ int remind;//device variable 

/* I am solving this problem in three stage
1.
In the first stage I am calculating the prefix sum over the facility array so that if I know the centre number and facility number'
we can get its maximum capacity in O(1) time. This is also used to get the number of request for any facility of any centre.
in my code.I am doing it via parallel prefix sum algorithm.

2.In the second stage I am sorting the array storing the request info so that i know the number of request for any facilty 
of any centre and I know the staring and last index of all the request corresponding to any facility of any centre.


3.in the third stage I am finally iterating over all the facility than serving its corresponding request if possible by launching kernel.
*/

//*******************************************

// Write down the kernels here




/*it is used to set the value of the device variable*/
__global__ void setvalue(int value){
    remind=value;
}




/* this function is used to calculate prefix sum of an array parallely. The maximum size of the array can be 1024 that is why 
we don't need a global barrier .Syncthreads will be sufficient.It takes as argument the array, size of the array over which
we want to calculate the prefix sum and also the startign index of the array
*/

__global__ void prefixSum(int *a,int a_size,int start){ 
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    int tempvar=0;
    if(tid==0)
        a[tid+start+1]+=remind;//Adding the last element of the previous block over which we calculated prefix summ to maintain consistency 
    __syncthreads();

    /*doing the obvious parallel prefix sum
    */
    for(int off=1;off<a_size;off*=2){
       
        if(tid>=off){
            tempvar=a[tid-off+start+1];
        }
        __syncthreads();
        if(tid>=off){
            a[tid+start+1]+=tempvar;
        }
        __syncthreads();
        
    }

    //again storing the last element into the device variable remind which acts as a remainder which is going to used when we 
    //calculate prefix sum of the next block
    if(tid==0){
        remind=a[start+a_size-1+1];
    } 
}






/*This is the main function and each instance of this kernel running on a thread represents a facility of a centre.It goes over all the
request of that facility and tries to accomodate that request according to maximum capacity of taht request.If request is granted
then increaments number of succesfull request for the corresponding centre
*/
__global__ void kernel1(int request,    int *d_req_id, int *d_req_cen, int *d_req_fac, int *d_req_start, int *d_req_slots,int startind,int lastind,int cen,int fac,int cap,int * d_succ_reqs  ){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<request){// check that it is valid request 
        int arr[24];//indicating capacity of each faciltiy at each timestamp initially it is original capacity of faciltiy
        for(int i=0;i<24;i++){
            arr[i]=cap;
        }
        
        //now accessing each request of that facility priority of smaller request id will be taken care of implicitly
        for(int j=startind;j<=lastind;j++){
            int start=d_req_start[j]-1;//start timestamp of that request
            int slot=d_req_slots[j];//number of slots for tha trequest
            int k;

            //for each request check availibilty of the request
            for(k=start;k<start+slot;k++){
                if(arr[k]==0){
                    //condition satisfies means request cant be granted
                    break;
                }
            }


            if(k==start+slot){//facility is available 
                atomicAdd(&d_succ_reqs[cen],1); //increamenting number of succesfull request for the corresponding centre                      
                for(int m=start;m<start+slot;m++){
                    arr[m]-=1;//decreamentign the capacity for that timestamp interval because is is occupies by the above successful request
                }
            }
        }
        
    }

}

//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N, *centre ,*facility ,*capacity ,*fac_ids , *succ_reqs , *tot_reqs ;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    memset(centre, 0, N*sizeof(int));
    
    /*This one is important Anup*/
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    memset(facility, 0, N*sizeof(int));

    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    memset(fac_ids, 0, max_P * N*sizeof(int));

    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 
    memset(capacity, 0, max_P * N*sizeof(int));


 
    int total=0;
    int success=0;  // total successful requests
    int fail = 0;   // total failed requests


    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    memset(tot_reqs, 0, N*sizeof(int));

    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre
    memset(succ_reqs, 0, N*sizeof(int));


    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }



    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request

    // Allocate memory on CPU 
	int R;


  
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
        //code the kernels here
    //*******************************************************************************************************************************		


   ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
   //This is first stage of our overall solution that is finding prefix sum so that we can get the index of capacity array
   //when we know the centre number and facilty number so that we can get the corresponding maximum capacity in O(1) time
   /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////// 
   
   
    setvalue<<<1,1>>>(0);//setting the value of our device variable remind to be initailly zero it represnets remainder of the
    // previous block over which prefix sum is calculated
    hipDeviceSynchronize();


    /*d_newfacility is a device array of size N+1 .It is used to store the prefix sum of the array which stores the no. of facility
    of each centre. Our prefix sum will have its first element as zero. So that if we want to know the capacity of first facility
    ith centre we can get that by accessing index "d_newfacility[i]"" of capacity  array or if we want to access the second facility 
    of ith centre  we can get that by accessing index "d_newfacility[i]+1" of capacity  array ans so on
    */
    int *d_newfacility;
    hipMalloc(&d_newfacility, (N+1)*sizeof(int));
    hipMemset(d_newfacility, 0, (N+1)* sizeof(int));
    hipMemcpy(d_newfacility+1,facility, N*sizeof(int), hipMemcpyHostToDevice);

    /*we are deviding N that is no. of centres in blocks of size 1024 and then computing the prefix sum of each block one at a time
    after calculating for one block the last elelment of that block is stored in device  variable "remind" and then we are calculating
    the prefix sum of the next block and we are making sure that the remind variable is added appropriately so that we get the overall 
    prefix sum of the array and consistency is maintained.
    the reason I am calculating prefix sum over blocks of size 1024 because in that way I don't have to use Global barrier
    syncthreads will be sufficient
    */
    int p=((N)/1024);//number of blocks in our  each of size 1024
    int r=((N)%1024);//size of the last block 
    int starting=0;
    for(int i=0;i<p;i++){
         starting=i*1024;
         //calculating prefix sum of each block
         prefixSum<<<1,1024>>>(d_newfacility,1024,starting);//kernel takes device array,size of the block,and starting index of that block
         hipDeviceSynchronize();
     }
     starting=p*1024;//starting index of the last block
     prefixSum<<<1,r>>>(d_newfacility,r,starting);//calculating prefix sum of last block
    
    
    //copying the prefix sum to host array cumfacility array
    int *cumfacility=(int*)malloc((N+1) * sizeof (int));
    hipMemcpy(cumfacility,d_newfacility, (N+1)*sizeof(int), hipMemcpyDeviceToHost);

    
    /*Now this nor array is very important and its similar to the capacity array that is first each element represents a facility of 
    a centre smilar to how facility array stores the capacity of each facility of each centre this nor array stores the number of
    request of each facility of each centre .And one more thing similar to how we were accessing the capacity of any facility of any 
    centre similar to that we can access the number of request corresponding to any facility of any centre .Indexing used will be
     similar  by takign help of the prefix sum we calculated
    */
    int *nor=(int*)malloc(max_P * N * sizeof (int));
    memset(nor, 0, max_P * N * sizeof(int));




/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//This is the second stage of our overall solution that is sorting of the request array
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


/*In this step we are sorting our request array in such a way that a request requesting for a centre with smaller centre number
comes first and if centre number is same than a request requesting for a smaller facility comes first and at  last if a request 
is requestign for the same centre number and also the same facility number than the request with smaller request id should come
first ,according to question.

We are increamenting the number of request for each facility of each centre in the nor array also.
*/

    for(int i=0;i<R;i++){
      int id=req_id[i];
      int cen=req_cen[i];
      int fac=req_fac[i];
      int start=req_start[i];
      int slot=req_slots[i];


      nor[cumfacility[cen]+fac]+=1;//increamenting the number of request for that particukar facility

        int j;
      for( j=i-1;j>=0;j--){
        if( (req_cen[j]>cen) || (req_cen[j]==cen && req_fac[j]>fac )|| (req_cen[j]==cen && req_fac[j]==fac && req_id[j]>id ) ){
          req_id[j+1]=req_id[j];
          req_cen[j+1]=req_cen[j];
          req_fac[j+1]=req_fac[j];
          req_start[j+1]=req_start[j];
          req_slots[j+1]=req_slots[j];
        }
        else{
            break;
        }
      }
          req_id[j+1]=id;
          req_cen[j+1]=cen;
          req_fac[j+1]=fac;
          req_start[j+1]=start;
          req_slots[j+1]=slot;

    }

    int * d_succ_reqs;// decice array for number of successfull request for each centre
    hipMalloc(&d_succ_reqs, N*sizeof(int));
    hipMemset(d_succ_reqs, 0, N* sizeof(int));



/* 
 Here i am just transfering the request array info on to the device
*/
    int *d_req_id, *d_req_cen, *d_req_fac, *d_req_start, *d_req_slots;   
    hipMalloc(&d_req_id, R*sizeof(int));
    hipMalloc(&d_req_cen, R*sizeof(int));
    hipMalloc(&d_req_fac, R*sizeof(int));
    hipMalloc(&d_req_start, R*sizeof(int));
    hipMalloc(&d_req_slots, R*sizeof(int));
    hipMemcpy(d_req_id,req_id, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_cen,req_cen, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_fac,req_fac, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_start,req_start, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_slots,req_slots, R*sizeof(int), hipMemcpyHostToDevice);



    int totalfacility=cumfacility[N-1]+facility[N-1];//total no of facility in our problem
    int mycen=-1,myfac=-1,reqind=0;




//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//This is the third stage of our overall solution that is finally serving each request 
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




/*Now we are at the third stage of our solution. In this stage we are iterating over all the facility and considering
each request of that facility if there is any request for that facility.In each iteration of the loop variable represents the facility
variable mycen contains the corresponding centre number and myfac contains the corresponding request number and for 
each facility we have its number of request and we also have the starting index of the request and also the ending index of 
the request because we have sorted our request array in similar manner
*/
    for(int i=0;i<totalfacility;i++){
        int request=nor[i];// number of request for that facility
        if(fac_ids[i]==0){
            mycen+=1;
            myfac=0;
        }
        else{
            myfac+=1;
        }
        if(request==0)// if no request is for that facility than move on to the next facility
            continue;
       
        int mycapacity=capacity[cumfacility[mycen]+myfac];//maximum capacity of taht facility
        int startind=reqind;//in the request array starting request index of that facility
        int lastind=reqind+request-1;//in the request array last request index of tha facility
        /*launching a thread for each facility it takes as argumnet number of request for that facility all the request array info
        its center number so that if request is accepted it increaments the number of successful request in the d_succ_reqs array
        for that centre, it also takes facility number of that centre and also the maximum capacity of that facility 
        */
        kernel1<<<1,1>>>(request,   d_req_id,d_req_cen,d_req_fac,d_req_start,d_req_slots,startind,lastind,   mycen,myfac,mycapacity, d_succ_reqs);
        reqind+=request;
    }
    hipDeviceSynchronize();

    //copying  back number of successfull request for each centre back to host 
    hipMemcpy(succ_reqs,d_succ_reqs, N*sizeof(int), hipMemcpyDeviceToHost);
    int total_succ=0;

    /*counting the total number of request for each centre and also the total number of successfull request for each centre
    */
    for(int i=0;i<N;i++){
        total+=tot_reqs[i];
        total_succ+=succ_reqs[i];
        
    }    
    success=total_succ;
    fail=total-total_succ;// total number of failed request

    //****************************************************************************************************************************





    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}